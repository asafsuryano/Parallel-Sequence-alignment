#include "hip/hip_runtime.h"
//this is a an experiment project
extern "C"{
#include "cudaFuncsHeader.h"
#include "stringFuncs.h"
}
#include <stdio.h>
#include <string.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>


__device__ int checkIfInConservativeGroup(char ch1,char ch2);
__device__ int checkIfInSemiConservativeGroup(char ch1,char ch2);
__device__ int compareTwoChars(char ch1,char ch2);
__global__ void compareAndReturnResult(char* seq1,char* seq2,int numOfCharsInSeq2,int n,int hyphen_index,double co1,double co2,double co3,double co4,int* pair_value);
__global__ void resetCounters();


//counters that will be in global memory so that all blocks and threads will access the same counters
__device__ int num_of_stars;
__device__ int num_of_colons;
__device__ int num_of_dots;
__device__ int num_of_nothing;

//global variable for sequence 1 on the gpu
char* seq1_on_gpu;
//global variable for sequence 2 on the gpu
char* seq2_On_Gpu;
//global variables for the coefficients (they are on the host side)
double co1_con,co2_con,co3_con,co4_con;
// pair value
int* pair_value_gpu;


//allocate sequence 1 to the gpu and it will stay there until the end of the program
extern "C"
void allocateSeq1AndCoefficients(char* seq1,int numOfChars,double co1,double co2,double co3,double co4)
{
	//int* pair;
	hipMalloc((void**)&seq1_on_gpu,numOfChars);
	hipMemcpy(seq1_on_gpu,seq1,numOfChars,hipMemcpyHostToDevice);
	co1_con=co1;
	co2_con=co2;
	co3_con=co3;
	co4_con=co4;
	FILE* file=fopen("pair_values.txt","r");
	int* pair_value_arr=(int*)malloc(sizeof(int)*26*26);
	int i;
	for (i=0;i<26*26;i++)
	{
		fscanf(file,"%d",(pair_value_arr+i));
	}
	fclose(file);
	checkCudaErrors(hipMalloc((void**)&pair_value_gpu,sizeof(int)*26*26));
	//checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(pair_value_gpu),pair_value_arr,sizeof(int)*26*26,0,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pair_value_gpu,pair_value_arr,sizeof(int)*26*26,hipMemcpyHostToDevice));
}

extern "C"
void allocateSeq2(char* seq2Private,int numOfChars)
{
	//allocate space in gpu for sequence 2
	checkCudaErrors(hipMalloc((void**)&seq2_On_Gpu,(numOfChars+1)*sizeof(char)));
	//copy the contents from sequence 2 to sequence 2 on the gpu
	checkCudaErrors(hipMemcpy(seq2_On_Gpu,seq2Private,numOfChars+1,hipMemcpyHostToDevice));
}
// this function will calculate the score with the help of cuda
extern "C"
double startCuda(int numOfChars,int i,int k,int seqNum)
{
	resetCounters<<<1,1>>>();
	int stars=0,colons=0,dots=0,nothing=0;
	int threads_per_block=256;
	int blocks_per_grid=(numOfChars+threads_per_block-1)/threads_per_block;
	//call the kernel function that will count the number of stars,colons,dots and nothing
	compareAndReturnResult<<<blocks_per_grid,threads_per_block>>>(seq1_on_gpu,seq2_On_Gpu,numOfChars,i,k,co1_con,co2_con,co3_con,co4_con,pair_value_gpu);
	//get the values from the counters in global memory
	copyCountersFromCudaMemToNormalMem(&stars,&colons,&dots,&nothing);
	//calculate and return the score
	double score=stars*co1_con-colons*co2_con-dots*co3_con-nothing*co4_con;
	return score;

}

extern "C"
void freeSeq2()
{
	checkCudaErrors(hipFree(seq2_On_Gpu));
}

void copyCountersFromCudaMemToNormalMem(int* stars,int* colons,int* dots,int* nothing)
{
	checkCudaErrors(hipMemcpyFromSymbol(stars,HIP_SYMBOL(num_of_stars),sizeof(int),0,hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpyFromSymbol(colons,HIP_SYMBOL(num_of_colons),sizeof(int),0,hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpyFromSymbol(dots,HIP_SYMBOL(num_of_dots),sizeof(int),0,hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpyFromSymbol(nothing,HIP_SYMBOL(num_of_nothing),sizeof(int),0,hipMemcpyDeviceToHost));
}

//returns 1 if two chars are identical
//returns 2 if two are in a conservative group
// returns 3 if two chars are in a semi conservative group
// returns 4 if two chars are none of the above
__device__ int compareTwoChars(char ch1,char ch2)
{
	
	if (ch1==ch2)
	{
		return 1;
	}
	else if (checkIfInConservativeGroup(ch1,ch2)==1)
	{
		return 2;
	}
	else if (checkIfInSemiConservativeGroup(ch1,ch2)==1)
	{
		return 3;
	}
	else
	{
		return 4;
	}
}

//this function will return 1 if they are in a conservative group or else 0
__device__ int checkIfInConservativeGroup(char ch1,char ch2)
{
	int numOfGroups=9;
	//define the conservative group
	const char *conservative[]={"NDEQ\0","NEQK\0","STA\0","MILV\0","QHRK\0","NHQK\0","FYW\0","HY\0","MILF\0"};
	//define the number of characters in each group
	const int conservative_num[]={4,4,3,4,4,4,3,2,4};
	int i,j;
	int ch1_is_in=0,ch2_is_in=0;
	for (i=0;i<numOfGroups;i++)
	{
		for (j=0;j<conservative_num[i];j++)
		{
			if (ch1==conservative[i][j])
			{
				ch1_is_in=1;
			}
			if (ch2==conservative[i][j])
			{
				ch2_is_in=1;
			}
		}
		if ((ch1_is_in==1)&&(ch2_is_in==1))
		{
			return 1;
		}
		ch1_is_in=0;
		ch2_is_in=0;
	}
	return 0;
}

//this function will return 1 if they are in a semi conservative group or else 0
__device__ int checkIfInSemiConservativeGroup(char ch1,char ch2)
{
	int numOfGroups=11;
	//define the semi conservative groups
	const char* semiConservative[]={"SAG\0","ATV\0","CSA\0","SGND\0","STPA\0","STNK\0","NEQHRK\0","NDEQHK\0","SNDEQK\0","HFY\0","FVLIM\0"};
	//define the number of characters in each group
	const int semi_conservative_num[]={3,3,3,4,4,4,6,6,6,3,5};
	int i,j;
	int ch1_is_in=0,ch2_is_in=0;
	for (i=0;i<numOfGroups;i++)
	{
		for (j=0;j<semi_conservative_num[i];j++)
		{
			if (ch1==semiConservative[i][j])
			{
				ch1_is_in=1;
			}
			if (ch2==semiConservative[i][j])
			{
				ch2_is_in=1;
			}
		}
		if ((ch1_is_in==1)&&(ch2_is_in==1))
		{
			return 1;
		}
		ch1_is_in=0;
		ch2_is_in=0;
	}
	return 0;
}


//this function compares seq1 to seq2 given offset n and returns the score
__global__ void compareAndReturnResult(char* seq1,char* seq2,int numOfCharsInSeq2,int n,int hyphen_index,double co1,double co2,double co3,double co4,int* pair_value)
{
	int index,ans=0;
	char ch1,ch2;
	index=blockDim.x * blockIdx.x + threadIdx.x;
	if (index>=hyphen_index)
		ch1=seq1[n+index+1];
	else
		ch1=seq1[n+index];
	ch2=seq2[index];
	if (index<numOfCharsInSeq2)
	{

		ans = pair_value[(ch1-65)*26+(ch2-65)];
		switch(ans)
		{
		case 1:
			//printf("got a star\n");
			atomicAdd(&num_of_stars,1);
			break;
		case 2:
			atomicAdd(&num_of_colons,1);
			break;
		case 3:
			atomicAdd(&num_of_dots,1);
			break;
		case 4:
			atomicAdd(&num_of_nothing,1);
			break;
		default:
			break;
		}
	}
}

//reset all counters
__global__ void resetCounters()
{
	 num_of_stars=0;
	 num_of_colons=0;
	 num_of_dots=0;
	 num_of_nothing=0;
}





